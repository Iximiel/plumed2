#include "hip/hip_runtime.h"
/* +++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
   Copyright (c) 2011-2023 The plumed team
   (see the PEOPLE file at the root of the distribution for a list of names)

   See http://www.plumed.org for more information.

   This file is part of plumed, version 2.

   plumed is free software: you can redistribute it and/or modify
   it under the terms of the GNU Lesser General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   plumed is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with plumed.  If not, see <http://www.gnu.org/licenses/>.
+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
#include "plumed/colvar/CoordinationBase.h"
#include "plumed/tools/SwitchingFunction.h"
#include "plumed/tools/NeighborList.h"
#include "plumed/core/ActionRegister.h"

#include "ndReduction.h"

#include "hip/hip_runtime.h"
#include ""

#include <numeric>
#include <limits>
#include <iostream>

using std::cerr;

#define vdbg(...) std::cerr << __LINE__ <<":" << #__VA_ARGS__ << " " << (__VA_ARGS__) <<'\n'
//#define vdbg(...)

namespace PLMD {
namespace colvar {

//+PLUMEDOC COLVAR CUDACOORDINATION
/*
Calculate coordination numbers. Like coordination, but on nvdia gpu and with no swithcing function

This keyword can be used to calculate the number of contacts between two groups of atoms
and is defined as
\f[
\sum_{i\in A} \sum_{i\in B} s_{ij}
\f]
where \f$s_{ij}\f$ is 1 if the contact between atoms \f$i\f$ and \f$j\f$ is formed,
zero otherwise.
In actuality, \f$s_{ij}\f$ is replaced with a switching function so as to ensure that the calculated CV has continuous derivatives.
The default switching function is:
\f[
s_{ij} = \frac{ 1 - \left(\frac{{\bf r}_{ij}-d_0}{r_0}\right)^n } { 1 - \left(\frac{{\bf r}_{ij}-d_0}{r_0}\right)^m }
\f]
but it can be changed using the optional SWITCH option.

To make your calculation faster you can use a neighbor list, which makes it that only a
relevant subset of the pairwise distance are calculated at every step.

If GROUPB is empty, it will sum the \f$\frac{N(N-1)}{2}\f$ pairs in GROUPA. This avoids computing
twice permuted indexes (e.g. pair (i,j) and (j,i)) thus running at twice the speed.

Notice that if there are common atoms between GROUPA and GROUPB the switching function should be
equal to one. These "self contacts" are discarded by plumed (since version 2.1),
so that they actually count as "zero".


\par Examples

The following example instructs plumed to calculate the total coordination number of the atoms in group 1-10 with the atoms in group 20-100.  For atoms 1-10 coordination numbers are calculated that count the number of atoms from the second group that are within 0.3 nm of the central atom.  A neighbor list is used to make this calculation faster, this neighbor list is updated every 100 steps.
\plumedfile
COORDINATION GROUPA=1-10 GROUPB=20-100 R_0=0.3 NLIST NL_CUTOFF=0.5 NL_STRIDE=100
\endplumedfile

The following is a dummy example which should compute the value 0 because the self interaction
of atom 1 is skipped. Notice that in plumed 2.0 "self interactions" were not skipped, and the
same calculation should return 1.
\plumedfile
c: COORDINATION GROUPA=1 GROUPB=1 R_0=0.3
PRINT ARG=c STRIDE=10
\endplumedfile

Here's an example that shows what happens when providing COORDINATION with
a single group:
\plumedfile
# define some huge group:
group: GROUP ATOMS=1-1000
# Here's coordination of a group against itself:
c1: COORDINATION GROUPA=group GROUPB=group R_0=0.3
# Here's coordination within a single group:
x: COORDINATION GROUPA=group R_0=0.3
# This is just multiplying times 2 the variable x:
c2: COMBINE ARG=x COEFFICIENTS=2 PERIODIC=NO

# the two variables c1 and c2 should be identical, but the calculation of c2 is twice faster
# since it runs on half of the pairs.
PRINT ARG=c1,c2 STRIDE=10
\endplumedfile



*/
//+ENDPLUMEDOC


//these constant will be used within the kernels
struct rationalSwitchParameters{
  double dmaxSQ=std::numeric_limits<double>::max();
  double invr0_2=1.0;//r0=1
  double stretch=1.0;
  double shift=0.0;
  int nn=6;
  int mm=12;
};

//does not inherit from coordination base because nl is private
class CudaCoordination : public Colvar {
  std::unique_ptr<NeighborList> nl;
  ///the pointer to the coordinates on the GPU
  double *cudaCoords;
  ///the pointer to the nn list on the GPU
  unsigned *cudaPairList;
  SwitchingFunction switchingFunction;
  rationalSwitchParameters switchingParameters;

  bool pbc{true};
  bool serial{false};
  bool invalidateList{true};
  bool firsttime{true};
  void setUpPermanentGPUMemory();
public:
  explicit CudaCoordination(const ActionOptions&);
  virtual ~CudaCoordination();
// active methods:
  static void registerKeywords( Keywords& keys );
  void prepare() override;
  void calculate() override;
};

PLUMED_REGISTER_ACTION(CudaCoordination,"CUDACOORDINATION")

void CudaCoordination::setUpPermanentGPUMemory(){
  auto nat = getPositions().size();
  hipFree(cudaCoords);
  hipFree(cudaPairList);
  //coordinates values are updated at each step
  hipMalloc(&cudaCoords, 3 * nat * sizeof(double));
  //the neighbour list will be updated at each request of prepare
  auto pairList = nl->getClosePairs();
  const unsigned nn=nl->size();
  hipMalloc(&cudaPairList, 2*nn*sizeof(unsigned));
  hipMemcpy(cudaPairList, pairList.data(),
            2*nn* sizeof(unsigned),
            hipMemcpyHostToDevice);
}

void CudaCoordination::prepare() {
  if(nl->getStride()>0) {
    if(firsttime || (getStep()%nl->getStride()==0)) {
      requestAtoms(nl->getFullAtomList());
      setUpPermanentGPUMemory();
      invalidateList=true;
      firsttime=false;
    } else {
      requestAtoms(nl->getReducedAtomList());
      setUpPermanentGPUMemory();
      invalidateList=false;
      if(getExchangeStep()) error("Neighbor lists should be updated on exchange steps - choose a NL_STRIDE which divides the exchange stride!");
    }
    if(getExchangeStep()) firsttime=true;
  }
}
void CudaCoordination::registerKeywords( Keywords& keys ) {
  Colvar::registerKeywords(keys);
  keys.addFlag("SERIAL",false,"Perform the calculation in serial - for debug purpose");
  keys.addFlag("PAIR",false,"Pair only 1st element of the 1st group with 1st element in the second, etc");
  keys.addFlag("NLIST",false,"Use a neighbor list to speed up the calculation");
  keys.add("optional","NL_CUTOFF","The cutoff for the neighbor list");
  keys.add("optional","NL_STRIDE","The frequency with which we are updating the atoms in the neighbor list");
  keys.add("atoms","GROUPA","First list of atoms");
  keys.add("atoms","GROUPB","Second list of atoms (if empty, N*(N-1)/2 pairs in GROUPA are counted)");
  keys.add("compulsory","NN","6","The n parameter of the switching function ");
  keys.add("compulsory","MM","0","The m parameter of the switching function; 0 implies 2*NN");
  keys.add("compulsory","D_0","0.0","The d_0 parameter of the switching function");
  keys.add("compulsory","R_0","The r_0 parameter of the switching function");
}

//these constant will be used within the kernels
__constant__ double cu_epsilon;

__device__ double pcuda_fastpow(double base,int expo) {
  if(expo<0) {
    expo=-expo;
    base=1.0/base;
  }
  double result = 1.0;
  while (expo) {
    if (expo & 1) {
      result *= base;
    }
    expo >>= 1;
    base *= base;
  }
  return result;
}

__device__ double pcuda_Rational(double rdist,double&dfunc,int NN, int MM) {
  double result;
  if(2*NN==MM) {
// if 2*N==M, then (1.0-rdist^N)/(1.0-rdist^M) = 1.0/(1.0+rdist^N)
    double rNdist=pcuda_fastpow(rdist,NN-1);
    double iden=1.0/(1+rNdist*rdist);
    dfunc = -NN*rNdist*iden*iden;
    result = iden;
  } else {
    if(rdist>(1.-100.0*cu_epsilon) && rdist<(1+100.0*cu_epsilon)) {
      result=NN/MM;
      dfunc=0.5*NN*(NN-MM)/MM;
    } else {
      double rNdist=pcuda_fastpow(rdist,NN-1);
      double rMdist=pcuda_fastpow(rdist,MM-1);
      double num = 1.-rNdist*rdist;
      double iden = 1.0/(1.0-rMdist*rdist);
      double func = num*iden;
      result = func;
      dfunc = ((-NN*rNdist*iden)+(func*(iden*MM)*rMdist));
    }
  }
  return result;
}

__global__ void getpcuda_Rational(double *rdists,double *dfunc,int NN, int MM,
    double*res) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(rdists[i]<=0.) {
    res[i]=1.;
    dfunc[i]=0.0;
  }else{
  res[i]=pcuda_Rational(rdists[i],dfunc[i],NN,MM);
  }
  printf("CUDA: %i :: d=%f -> %f, %f\n", i,rdists[i],res[i],dfunc[i]);
}


__global__ void getConst() {
  printf("Cuda: cu_epsilon = %f\n", cu_epsilon);
}

CudaCoordination::CudaCoordination(const ActionOptions&ao):
  PLUMED_COLVAR_INIT(ao)

{
  parseFlag("SERIAL",serial);

  std::vector<AtomNumber> ga_lista,gb_lista;
  parseAtomList("GROUPA",ga_lista);
  parseAtomList("GROUPB",gb_lista);

  bool nopbc=!pbc;
  parseFlag("NOPBC",nopbc);
  pbc=!nopbc;

// pair stuff
  bool dopair=false;
  parseFlag("PAIR",dopair);

// neighbor list stuff
  bool doneigh=false;
  double nl_cut=0.0;
  int nl_st=0;
  parseFlag("NLIST",doneigh);
  if(doneigh) {
    parse("NL_CUTOFF",nl_cut);
    if(nl_cut<=0.0) error("NL_CUTOFF should be explicitly specified and positive");
    parse("NL_STRIDE",nl_st);
    if(nl_st<=0) error("NL_STRIDE should be explicitly specified and positive");
  }

  addValueWithDerivatives(); setNotPeriodic();
  if(gb_lista.size()>0) {
    if(doneigh)
    {nl=Tools::make_unique<NeighborList>(ga_lista,gb_lista,serial,dopair,pbc,getPbc(),comm,nl_cut,nl_st);}
    else
    {nl=Tools::make_unique<NeighborList>(ga_lista,gb_lista,serial,dopair,pbc,getPbc(),comm);}
  } else {
    if(doneigh)
    {nl=Tools::make_unique<NeighborList>(ga_lista,serial,pbc,getPbc(),comm,nl_cut,nl_st);}
    else
    {nl=Tools::make_unique<NeighborList>(ga_lista,serial,pbc,getPbc(),comm);}
  }

  requestAtoms(nl->getFullAtomList());

  log.printf("  between two groups of %u and %u atoms\n",static_cast<unsigned>(ga_lista.size()),static_cast<unsigned>(gb_lista.size()));
  log.printf("  first group:\n");
  for(unsigned int i=0; i<ga_lista.size(); ++i) {
    if ( (i+1) % 25 == 0 ) log.printf("  \n");
    log.printf("  %d", ga_lista[i].serial());
  }
  log.printf("  \n  second group:\n");
  for(unsigned int i=0; i<gb_lista.size(); ++i) {
    if ( (i+1) % 25 == 0 ) log.printf("  \n");
    log.printf("  %d", gb_lista[i].serial());
  }
  log.printf("  \n");
  if(pbc) log.printf("  using periodic boundary conditions\n");
  else    log.printf("  without periodic boundary conditions\n");
  if(dopair) log.printf("  with PAIR option\n");
  if(doneigh) {
    log.printf("  using neighbor lists with\n");
    log.printf("  update every %d steps and cutoff %f\n",nl_st,nl_cut);
  }
  std::string sw,errors;

  {//loading data to the GPU
    int nn_=6;
    int mm_=0;
    double d0_=0.0;
    double r0_=0.0;
    parse("R_0",r0_);
    if(r0_<=0.0) {
      error("R_0 should be explicitly specified and positive");
    }
    parse("D_0",d0_);
    parse("NN",nn_);
    parse("MM",mm_);
    if(mm_==0) {
      mm_=2*nn_;
      }
      
    switchingParameters.nn=nn_;
    switchingParameters.mm=mm_;
    switchingParameters.stretch=1.0;
    switchingParameters.shift=0.0;
    double dmax=d0_+r0_*std::pow(0.00001,1./(nn_-mm_));
    constexpr bool dostretch=true;
    if (dostretch){
      std::vector<double> inputs = {0.0,dmax};
      double *inputsc,*dummy;
      double *sc;
      hipMalloc(&inputsc, 2 *sizeof(double));
      hipMalloc(&dummy, 2*sizeof(double));
      hipMalloc(&sc, 2*sizeof(double));
      hipMemcpy(inputsc, inputs.data(), 2* sizeof(double),
                hipMemcpyHostToDevice);
      getpcuda_Rational<<<1,2>>>(inputsc,dummy,nn_,mm_,sc);
      std::vector<double> s = {0.0,0.0};
      hipMemcpy(s.data(), sc, 2* sizeof(double),
                hipMemcpyDeviceToHost);
      hipFree(inputsc);
      hipFree(dummy);
      hipFree(sc);
      switchingParameters.stretch=1.0/(s[0]-s[1]);
      switchingParameters.shift=-s[1]*switchingParameters.stretch;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(cu_epsilon), &epsilon, sizeof(double));
    switchingParameters.dmaxSQ = dmax* dmax;
    double invr0 = 1.0/r0_;
    switchingParameters.invr0_2 = invr0*=invr0;
  }
  checkRead();
  setUpPermanentGPUMemory();
  log<<"  contacts are counted with cutoff "<<switchingFunction.description()<<"\n";
}

CudaCoordination::~CudaCoordination(){
  hipFree(cudaCoords);
  hipFree(cudaPairList);
}

__device__ double calculateSqr(double distancesq, double& dfunc , rationalSwitchParameters switchingParameters) {
  double result=0.0;
  dfunc=0.0;
  if(distancesq<switchingParameters.dmaxSQ) {
    const double rdist_2 = distancesq*switchingParameters.invr0_2;
    result=pcuda_Rational(rdist_2,dfunc,switchingParameters.nn/2,switchingParameters.mm/2);
    // chain rule:
    dfunc*=2*switchingParameters.invr0_2;
    // cu_stretch:
    result=result*switchingParameters.stretch+switchingParameters.shift;
    dfunc*=switchingParameters.stretch;
  }
  //printf("%f\n",result);
  return result;
}

#define X(I) 3*I
#define Y(I) 3*I+1
#define Z(I) 3*I+2
//
__global__ void getCoord(
                        const unsigned numOfPairs,
                        unsigned nat,
                        rationalSwitchParameters switchingParameters,
                        double *coordinates,
                        unsigned *pairList,
                        double *ncoordOut,
                        double *derivativeOut,
                        double *virialOut
                        ) {
  //blockDIm are the number of threads in your block
  const unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >=numOfPairs) {
    return;
  }
  const unsigned i0= pairList[i*2];
  const unsigned i1= pairList[i*2+1]; 
  if (i0 == i1) {
    return;
  }
  double d[3]={
    coordinates[X(i1)] - coordinates[X(i0)],
    coordinates[Y(i1)] - coordinates[Y(i0)],
    coordinates[Z(i1)] - coordinates[Z(i0)]
  };

  double dsq=(d[0] * d[0] + d[1] * d[1] + d[2] * d[2]);
  double dfunc=0.;
  ncoordOut[i]= calculateSqr(dsq,dfunc,switchingParameters);
  
  double dd[3] = {
    d[0]*dfunc,
    d[1]*dfunc,
    d[2]*dfunc
    };

  nat*=i*3;
  //this needs a barrier!
  derivativeOut[X(i0)+nat] -=dd[0];
  derivativeOut[Y(i0)+nat] -=dd[1];
  derivativeOut[Z(i0)+nat] -=dd[2];

  derivativeOut[X(i1)+nat] +=dd[0];
  derivativeOut[Y(i1)+nat] +=dd[1];
  derivativeOut[Z(i1)+nat] +=dd[2];
  
  for(unsigned ii=0; ii<3; ++ii){
    for(unsigned jj=0; jj<3; ++jj){
      virialOut[9*i + ii*3+jj]=-dd[ii]*d[jj];
    }
  }
  

  //printf("Cuda:[%i]->%f\n",i,ncoord[i]);
  //ncoord[i]= 1;
  //printf("Cuda: %i,%i %i %i\n", i,threadIdx.x , blockIdx.x,  blockDim.x);
}

void CudaCoordination::calculate() {
  
  auto positions = getPositions();
  auto nat = positions.size();
  if(nl->getStride()>0 && invalidateList) {
    nl->update(getPositions());
  }
  auto pairList = nl->getClosePairs();
  const unsigned nn=nl->size();
  
  constexpr unsigned nthreads=256;
  // nextpw2 will be set up when the reduction will be done on the CPU
  //note nn shoudl be 1/2 pX($1)airList.size()
  //calculates the closest power of 2 (c++20 will have bit::bit_ceil(cu_nn))
  const size_t nnToGPU=nn;//pow(2, ceil(log2(nn)));
  //the occupancy MUST be set up correctly
  
  unsigned ngroups=ceil(double(nnToGPU)/nthreads);

  //donw here I am calling all the arrays that goes on the GPU cudaSomething

  /****************allocating the memory on the GPU****************/
  
  hipMemcpy(this->cudaCoords, &positions[0][0], 3 *nat* sizeof(double),
             hipMemcpyHostToDevice);
  double *cudaCoordination;
  hipMalloc(&cudaCoordination, nnToGPU * sizeof(double));
  
  double *cudaDev;
  hipMalloc(&cudaDev, nnToGPU *3*nat * sizeof(double));
  double *cudaVirial;
  hipMalloc(&cudaVirial, 9*nnToGPU * sizeof(double));
  /****************starting the calculations****************/
  getCoord<<<ngroups,nthreads>>> (nn,nat,switchingParameters,cudaCoords,cudaPairList,
    cudaCoordination,cudaDev,cudaVirial); 
  
  //std::vector<double> coordsToSUM(nn);
  //hipMemcpy(coordsToSUM.data(), cudaCoordination, nn*sizeof(double), hipMemcpyDeviceToHost);
  //double ncoord=std::accumulate(coordsToSUM.begin(),coordsToSUM.end(),0.0);
  double ncoord = CUDAHELPERS::reduceScalar(cudaCoordination, nn);

  Tensor virial=CUDAHELPERS::reduceTensor(cudaVirial, nn);
  std::vector<Vector> deriv = CUDAHELPERS::reduceNVectors(cudaDev,nn,nat);

  for(unsigned i=0; i<deriv.size(); ++i) {
    setAtomsDerivatives(i,deriv[i]);
  }
  hipFree(cudaCoordination);
  hipFree(cudaDev);
  hipFree(cudaVirial);

  setValue           (ncoord);
  setBoxDerivatives  (virial);
}

} // namespace colvar
} // namespace PLMD
