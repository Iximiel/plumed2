#include "hip/hip_runtime.h"
/* +++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
   Copyright (c) 2011-2023 The plumed team
   (see the PEOPLE file at the root of the distribution for a list of names)

   See http://www.plumed.org for more information.

   This file is part of plumed, version 2.

   plumed is free software: you can redistribute it and/or modify
   it under the terms of the GNU Lesser General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   plumed is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU Lesser General Public License for more details.

   You should have received a copy of the GNU Lesser General Public License
   along with plumed.  If not, see <http://www.gnu.org/licenses/>.
+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
#include "plumed/colvar/CoordinationBase.h"
#include "plumed/tools/SwitchingFunction.h"
#include "plumed/tools/NeighborList.h"
#include "plumed/core/ActionRegister.h"
#include "hip/hip_runtime.h"
#include ""
#include <numeric>
#include <iostream>
using std::cerr;

#define dbghere() cerr << __LINE__  << " "

namespace PLMD {
namespace colvar {

//+PLUMEDOC COLVAR CUDACOORDINATION
/*
Calculate coordination numbers. Like coordination, but on nvdia gpu and with no swithcing function

This keyword can be used to calculate the number of contacts between two groups of atoms
and is defined as
\f[
\sum_{i\in A} \sum_{i\in B} s_{ij}
\f]
where \f$s_{ij}\f$ is 1 if the contact between atoms \f$i\f$ and \f$j\f$ is formed,
zero otherwise.
In actuality, \f$s_{ij}\f$ is replaced with a switching function so as to ensure that the calculated CV has continuous derivatives.
The default switching function is:
\f[
s_{ij} = \frac{ 1 - \left(\frac{{\bf r}_{ij}-d_0}{r_0}\right)^n } { 1 - \left(\frac{{\bf r}_{ij}-d_0}{r_0}\right)^m }
\f]
but it can be changed using the optional SWITCH option.

To make your calculation faster you can use a neighbor list, which makes it that only a
relevant subset of the pairwise distance are calculated at every step.

If GROUPB is empty, it will sum the \f$\frac{N(N-1)}{2}\f$ pairs in GROUPA. This avoids computing
twice permuted indexes (e.g. pair (i,j) and (j,i)) thus running at twice the speed.

Notice that if there are common atoms between GROUPA and GROUPB the switching function should be
equal to one. These "self contacts" are discarded by plumed (since version 2.1),
so that they actually count as "zero".


\par Examples

The following example instructs plumed to calculate the total coordination number of the atoms in group 1-10 with the atoms in group 20-100.  For atoms 1-10 coordination numbers are calculated that count the number of atoms from the second group that are within 0.3 nm of the central atom.  A neighbor list is used to make this calculation faster, this neighbor list is updated every 100 steps.
\plumedfile
COORDINATION GROUPA=1-10 GROUPB=20-100 R_0=0.3 NLIST NL_CUTOFF=0.5 NL_STRIDE=100
\endplumedfile

The following is a dummy example which should compute the value 0 because the self interaction
of atom 1 is skipped. Notice that in plumed 2.0 "self interactions" were not skipped, and the
same calculation should return 1.
\plumedfile
c: COORDINATION GROUPA=1 GROUPB=1 R_0=0.3
PRINT ARG=c STRIDE=10
\endplumedfile

Here's an example that shows what happens when providing COORDINATION with
a single group:
\plumedfile
# define some huge group:
group: GROUP ATOMS=1-1000
# Here's coordination of a group against itself:
c1: COORDINATION GROUPA=group GROUPB=group R_0=0.3
# Here's coordination within a single group:
x: COORDINATION GROUPA=group R_0=0.3
# This is just multiplying times 2 the variable x:
c2: COMBINE ARG=x COEFFICIENTS=2 PERIODIC=NO

# the two variables c1 and c2 should be identical, but the calculation of c2 is twice faster
# since it runs on half of the pairs.
PRINT ARG=c1,c2 STRIDE=10
\endplumedfile



*/
//+ENDPLUMEDOC

//does not inherit from coordination base because nl is private
class CudaCoordination : public Colvar {
  bool pbc{true};
  bool serial{false};
  std::unique_ptr<NeighborList> nl;
  bool invalidateList{true};
  bool firsttime{true};
  SwitchingFunction switchingFunction;

public:
  explicit CudaCoordination(const ActionOptions&);
// active methods:
  static void registerKeywords( Keywords& keys );
  void prepare() override;
  void calculate() override;
};

PLUMED_REGISTER_ACTION(CudaCoordination,"CUDACOORDINATION")

void CudaCoordination::prepare() {
  if(nl->getStride()>0) {
    if(firsttime || (getStep()%nl->getStride()==0)) {
      requestAtoms(nl->getFullAtomList());
      invalidateList=true;
      firsttime=false;
    } else {
      requestAtoms(nl->getReducedAtomList());
      invalidateList=false;
      if(getExchangeStep()) error("Neighbor lists should be updated on exchange steps - choose a NL_STRIDE which divides the exchange stride!");
    }
    if(getExchangeStep()) firsttime=true;
  }
}
void CudaCoordination::registerKeywords( Keywords& keys ) {
  Colvar::registerKeywords(keys);
  keys.addFlag("SERIAL",false,"Perform the calculation in serial - for debug purpose");
  keys.addFlag("PAIR",false,"Pair only 1st element of the 1st group with 1st element in the second, etc");
  keys.addFlag("NLIST",false,"Use a neighbor list to speed up the calculation");
  keys.add("optional","NL_CUTOFF","The cutoff for the neighbor list");
  keys.add("optional","NL_STRIDE","The frequency with which we are updating the atoms in the neighbor list");
  keys.add("atoms","GROUPA","First list of atoms");
  keys.add("atoms","GROUPB","Second list of atoms (if empty, N*(N-1)/2 pairs in GROUPA are counted)");
  keys.add("compulsory","NN","6","The n parameter of the switching function ");
  keys.add("compulsory","MM","0","The m parameter of the switching function; 0 implies 2*NN");
  keys.add("compulsory","D_0","0.0","The d_0 parameter of the switching function");
  keys.add("compulsory","R_0","The r_0 parameter of the switching function");
}

//these constant will be used within the kernels
__constant__ double cu_dmaxSQ;
__constant__ double cu_invr0_2;
__constant__ double cu_stretch;
__constant__ double cu_shift;
__constant__ double cu_epsilon;
__constant__ int cu_nn;
__constant__ int cu_mm;

__device__ double pcuda_fastpow(double base,int expo) {
  if(expo<0) {
    expo=-expo;
    base=1.0/base;
  }
  double result = 1.0;
  while (expo) {
    if (expo & 1) {
      result *= base;
    }
    expo >>= 1;
    base *= base;
  }
  return result;
}

__device__ double pcuda_Rational(double rdist,double&dfunc,int NN, int MM) {
  double result;
  if(2*NN==MM) {
// if 2*N==M, then (1.0-rdist^N)/(1.0-rdist^M) = 1.0/(1.0+rdist^N)
    double rNdist=pcuda_fastpow(rdist,NN-1);
    double iden=1.0/(1+rNdist*rdist);
    dfunc = -NN*rNdist*iden*iden;
    result = iden;
  } else {
    if(rdist>(1.-100.0*cu_epsilon) && rdist<(1+100.0*cu_epsilon)) {
      result=NN/MM;
      dfunc=0.5*NN*(NN-MM)/MM;
    } else {
      double rNdist=pcuda_fastpow(rdist,NN-1);
      double rMdist=pcuda_fastpow(rdist,MM-1);
      double num = 1.-rNdist*rdist;
      double iden = 1.0/(1.0-rMdist*rdist);
      double func = num*iden;
      result = func;
      dfunc = ((-NN*rNdist*iden)+(func*(iden*MM)*rMdist));
    }
  }
  return result;
}

__global__ void getpcuda_Rational(double *rdists,double *dfunc,int NN, int MM,
    double*res) {
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(rdists[i]<=0.) {
    res[i]=1.;
    dfunc[i]=0.0;
  }else{
  res[i]=pcuda_Rational(rdists[i],dfunc[i],NN,MM);
  }
  printf("CUDA: %i :: d=%f -> %f, %f\n", i,rdists[i],res[i],dfunc[i]);
}


__global__ void getConst() {
  printf("Cuda: cu_dmaxSQ = %f\n", cu_dmaxSQ);
  printf("Cuda: cu_invr0_2 = %f\n", cu_invr0_2);
  printf("Cuda: cu_stretch = %f\n", cu_stretch);
  printf("Cuda: cu_epsilon = %f\n", cu_epsilon);
  printf("Cuda: cu_shift = %f\n", cu_shift);
  printf("Cuda: cu_nn = %i\n", cu_nn);
  printf("Cuda: cu_mm = %i\n", cu_mm);
}

CudaCoordination::CudaCoordination(const ActionOptions&ao):
  PLUMED_COLVAR_INIT(ao)

{
  parseFlag("SERIAL",serial);

  std::vector<AtomNumber> ga_lista,gb_lista;
  parseAtomList("GROUPA",ga_lista);
  parseAtomList("GROUPB",gb_lista);

  bool nopbc=!pbc;
  parseFlag("NOPBC",nopbc);
  pbc=!nopbc;

// pair stuff
  bool dopair=false;
  parseFlag("PAIR",dopair);

// neighbor list stuff
  bool doneigh=false;
  double nl_cut=0.0;
  int nl_st=0;
  parseFlag("NLIST",doneigh);
  if(doneigh) {
    parse("NL_CUTOFF",nl_cut);
    if(nl_cut<=0.0) error("NL_CUTOFF should be explicitly specified and positive");
    parse("NL_STRIDE",nl_st);
    if(nl_st<=0) error("NL_STRIDE should be explicitly specified and positive");
  }

  addValueWithDerivatives(); setNotPeriodic();
  if(gb_lista.size()>0) {
    if(doneigh)
    {nl=Tools::make_unique<NeighborList>(ga_lista,gb_lista,serial,dopair,pbc,getPbc(),comm,nl_cut,nl_st);}
    else
    {nl=Tools::make_unique<NeighborList>(ga_lista,gb_lista,serial,dopair,pbc,getPbc(),comm);}
  } else {
    if(doneigh)
    {nl=Tools::make_unique<NeighborList>(ga_lista,serial,pbc,getPbc(),comm,nl_cut,nl_st);}
    else
    {nl=Tools::make_unique<NeighborList>(ga_lista,serial,pbc,getPbc(),comm);}
  }

  requestAtoms(nl->getFullAtomList());

  log.printf("  between two groups of %u and %u atoms\n",static_cast<unsigned>(ga_lista.size()),static_cast<unsigned>(gb_lista.size()));
  log.printf("  first group:\n");
  for(unsigned int i=0; i<ga_lista.size(); ++i) {
    if ( (i+1) % 25 == 0 ) log.printf("  \n");
    log.printf("  %d", ga_lista[i].serial());
  }
  log.printf("  \n  second group:\n");
  for(unsigned int i=0; i<gb_lista.size(); ++i) {
    if ( (i+1) % 25 == 0 ) log.printf("  \n");
    log.printf("  %d", gb_lista[i].serial());
  }
  log.printf("  \n");
  if(pbc) log.printf("  using periodic boundary conditions\n");
  else    log.printf("  without periodic boundary conditions\n");
  if(dopair) log.printf("  with PAIR option\n");
  if(doneigh) {
    log.printf("  using neighbor lists with\n");
    log.printf("  update every %d steps and cutoff %f\n",nl_st,nl_cut);
  }
  std::string sw,errors;

  {
    int nn_=6;
    int mm_=0;
    double d0_=0.0;
    double r0_=0.0;
    parse("R_0",r0_);
    if(r0_<=0.0) {error("R_0 should be explicitly specified and positive");}
    parse("D_0",d0_);
    parse("NN",nn_);
    parse("MM",mm_);
    if(mm_==0) mm_=2*nn_;
    hipMemcpyToSymbol(HIP_SYMBOL(cu_nn), &nn_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(cu_mm), &mm_, sizeof(int));
    double stretch_=1.0;
    double shift_=0.0;
    double dmax=d0_+r0_*std::pow(0.00001,1./(nn_-mm_));
    constexpr bool dostretch=false;
    if (dostretch){
      std::vector<double> inputs = {0.0,dmax};
      double *inputsc,*dummy;
      double *sc;
      hipMalloc(&inputsc, 2 *sizeof(double));
      hipMalloc(&dummy, 2*sizeof(double));
      hipMalloc(&sc, 2*sizeof(double));
      hipMemcpy(inputsc, inputs.data(), 2* sizeof(double),
                hipMemcpyHostToDevice);
      getpcuda_Rational<<<1,2>>>(inputsc,dummy,nn_,mm_,sc);
      std::vector<double> s = {0.0,0.0};
      hipMemcpy(s.data(), sc, 2* sizeof(double),
                hipMemcpyDeviceToHost);
      dbghere() 
      <<"I: "<<inputs[0]<<" "<<inputs[1]
      <<"\nO: "<<s[0]<<" "<<s[1]
      <<"\n";
      hipFree(inputsc);
      hipFree(dummy);
      hipFree(sc);
      stretch_=1.0/(s[0]-s[1]);
      shift_=-s[1]*cu_stretch;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(cu_stretch), &stretch_, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(cu_shift), &shift_, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(cu_epsilon), &epsilon, sizeof(double));

    dmax*=dmax;
    hipMemcpyToSymbol(HIP_SYMBOL(cu_dmaxSQ), &dmax, sizeof(double));
    double invr0_2 = 1.0/r0_;
    invr0_2*=invr0_2;
    hipMemcpyToSymbol(HIP_SYMBOL(cu_invr0_2), &invr0_2, sizeof(double));
  }
  checkRead();
   getConst<<<1,1>>>();
  log<<"  contacts are counted with cutoff "<<switchingFunction.description()<<"\n";
}

__device__ double calculateSqr(double distancesq, double& dfunc) {
  double result=0.0;
  dfunc=0.0;
  if(distancesq<cu_dmaxSQ) {
    const double rdist_2 = distancesq*cu_invr0_2;
    result=pcuda_Rational(rdist_2,dfunc,cu_nn/2,cu_mm/2);
    // chain rule:
    dfunc*=2*cu_invr0_2;
    // cu_stretch:
    result=result*cu_stretch+cu_shift;
    dfunc*=cu_stretch;
  }
  //printf("%f\n",result);
  return result;
}

__global__ void getCoord(double *ncoord,double *coordinates, unsigned *pairList,
                         unsigned numOfPairs) {
  //blockDIm are the number of threads in your block
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >=numOfPairs) {
    //printf("Cuda: return i=%i>=%i\n", i,numOfPairs);
    return;
  }
  unsigned i0= pairList[i*2];
  unsigned i1= pairList[i*2+1]; 
  //printf("Cuda: %i,%i %i %i\n", i0 ,i1, i ,numOfPairs);
  if (i0 == i1) {
    //printf("Cuda: return i0=%i i1=%i\n", i0 ,i1);
    return;
  }
  double dx = coordinates[3 * i0/**/] - coordinates[3 * i1/**/];
  double dy = coordinates[3 * i0 + 1] - coordinates[3 * i1 + 1];
  double dz = coordinates[3 * i0 + 2] - coordinates[3 * i1 + 2];

  double dsq=(dx * dx + dy * dy + dz * dz);
  double dfunc=0.;
  ncoord[i]= calculateSqr(dsq,dfunc);
  //printf("Cuda:[%i]->%f\n",i,ncoord[i]);
  //ncoord[i]= 1;
  //printf("Cuda: %i,%i %i %i\n", i,threadIdx.x , blockIdx.x,  blockDim.x);
}

void CudaCoordination::calculate() {
  Tensor virial;
  std::vector<Vector> deriv(getNumberOfAtoms());
  auto positions = getPositions();
  auto nat = positions.size();
  if(nl->getStride()>0 && invalidateList) {
    nl->update(getPositions());
  }
  auto pairList = nl->getClosePairs();
  const unsigned nn=nl->size();
  //note cu_nn shoudl be 1/2 pairList.size()

  //calculates the closest power of 2 (c++20 will have bit::bit_ceil(cu_nn))
  size_t nextpw2 = pow(2, ceil(log2(nn)));
  //the occupancy MUST be set up correctly
  constexpr unsigned nthreads=256;
  unsigned ngroups=ceil(double(nextpw2)/nthreads);
  //std::cerr <<cu_nn << " " <<pairList.size() << " " <<nextpw2<<"<\n";

  /****************allocating the memory on the GPU****************/
  double *coords;
  double *ncoords;
  unsigned *cudaPairList;
  hipMalloc(&coords, 3 * nat * sizeof(double));
  hipMemcpy(coords, &positions[0][0], 3 *nat* sizeof(double),
             hipMemcpyHostToDevice);
  hipMalloc(&ncoords, nn * sizeof(double));
  hipMalloc(&cudaPairList, 2*nn * sizeof(unsigned));
  //resizing the pairlist should not be necessary
  //pairList.resize(2*nextpw2);
  hipMemcpy(cudaPairList, pairList.data(), 2*nn* sizeof(unsigned),
             hipMemcpyHostToDevice);
  
  /****************starting the calculations****************/
  //getCoord<<<ngroups,nthreads>>> (ncoords,coords, cudaPairList,nn);
   getCoord<<<ngroups,nthreads>>> (ncoords,coords, cudaPairList,
   
   nn
   );
  
  std::vector<double> coordsToSUM(nn);
  hipMemcpy(coordsToSUM.data(), ncoords, nn*sizeof(double), hipMemcpyDeviceToHost);
  double ncoord=std::accumulate(coordsToSUM.begin(),coordsToSUM.end(),0.0);
  for(unsigned i=0; i<deriv.size(); ++i) {
    setAtomsDerivatives(i,deriv[i]);
  }
  setValue           (ncoord);
  setBoxDerivatives  (virial);
}

} // namespace colvar
} // namespace PLMD
