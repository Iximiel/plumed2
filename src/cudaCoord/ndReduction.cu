#include "hip/hip_runtime.h"
#include "ndReduction.h"

#include "hip/hip_runtime.h"
#include ""

#include <iomanip>
#include <iostream>
#include <vector>
#include <numeric>

//#define vdbg(...) std::cerr << std::setw(4) << __LINE__ <<":" << std::setw(20)<< #__VA_ARGS__ << " " << (__VA_ARGS__) <<'\n'
#define vdbg(...)

// * Grids map to GPUs
// * Blocks map to the MultiProcessors (MP)
// * Threads map to Stream Processors (SP)
// * Warps are groups of (32) threads that execute simultaneously


//There are a LOTS of unrolled loop down here,
//see this https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf to undestand why

//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dim3
//7.3.2. dim3
//This type is an integer vector type based on uint3 that is used to specify dimensions. 

namespace PLMD {
namespace CUDAHELPERS {


template <unsigned numThreads, typename T>
__device__ void warpReduce(volatile T* sdata, unsigned int place){
    if(numThreads >= 64){//compile time
      sdata[place] += sdata[place + 32];
    }
    if(numThreads >= 32){//compile time
      sdata[place] += sdata[place + 16];
    }
    if(numThreads >= 16){//compile time
      sdata[place] += sdata[place + 8];
    }
    if(numThreads >= 8){//compile time
      sdata[place] += sdata[place + 4];
    }
    if(numThreads >= 4){//compile time
      sdata[place] += sdata[place + 2];
    }
    if(numThreads >= 2){//compile time
      sdata[place] += sdata[place + 1];
    }
}

template <unsigned numThreads, typename T>
__global__ void reductionND(const T *g_idata, T *g_odata, const unsigned int len) {
  //playing with this 
  //https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
  auto sdata = shared_memory_proxy<T>();
  const unsigned int coord = blockIdx.y;
  const unsigned int place = threadIdx.x;
  // each thread loads one element from global to shared memory
  const unsigned int diplacement = blockIdx.y*len;
  unsigned int i = (numThreads*2)*blockIdx.x + place + diplacement;
  const unsigned int gridSize = (numThreads*2)*gridDim.x;
  const unsigned int trgt=len+diplacement;

  sdata[threadIdx.x] = T(0);
  while (i+numThreads < trgt) {
    sdata[threadIdx.x] += g_idata[i] + g_idata[i+numThreads];
    i+=gridSize;
  }
  while (i < trgt) {
    sdata[threadIdx.x] += g_idata[i];
     i+=gridSize;
  }

  __syncthreads();
  // do reduction in shared memory
  
  if (numThreads >= 512) {//compile time
    if (threadIdx.x  < 256) {
       sdata[threadIdx.x] += sdata[threadIdx.x + 256]; } __syncthreads(); 
    }
  if (numThreads >= 256) {//compile time
    if (threadIdx.x  < 128) {
       sdata[threadIdx.x] += sdata[threadIdx.x + 128]; } __syncthreads(); 
    }
  if (numThreads >= 128) {//compile time
    if (threadIdx. x < 64) { 
      sdata[threadIdx.x] += sdata[threadIdx.x + 64]; } __syncthreads();
    }
  //Instructions are SIMD synchronous within a warp
  //so no need for __syncthreads(), in the last iterations
  if (threadIdx.x < mymin(32u,numThreads/2)) {
    warpReduce<numThreads>(sdata, threadIdx.x);
  }
  // write result for this block to global memory
  if (threadIdx.x == 0){
    g_odata[blockIdx.x+blockIdx.y*gridDim.x] = sdata[0];    
  }
}

template <unsigned numThreads, typename T>
__global__ void reduction1D(T *g_idata, T *g_odata, const unsigned int len) {
  //extern __shared__ T sdata[numThreads];
  auto sdata = shared_memory_proxy<T>();
  const unsigned int place = threadIdx.x;
  // each thread loads one element from global to shared mem
  unsigned int i = numThreads*blockIdx.x*2 + place;
  const unsigned int gridSize = numThreads*gridDim.x*2;
  sdata[place] = T(0);
  //I think this may slow down the loop, but this does not force the user to have
  //an input that is multiple of the threads, padded with zeros
  while (i+numThreads < len) {
    sdata[place] += g_idata[i] + g_idata[i+numThreads];
    i+=gridSize;
  }
  while (i < len) {
    sdata[place] += g_idata[i];
    i+=gridSize;
  }
    
  __syncthreads();
  // do reduction in shared memory
  
  if (numThreads >= 512) {//compile time
    if (threadIdx.x  < 256) {
       sdata[place] += sdata[place + 256]; } __syncthreads(); 
       }
  if (numThreads >= 256) {//compile time
    if (threadIdx.x  < 128) {
       sdata[place] += sdata[place + 128]; } __syncthreads(); 
       }
  if (numThreads >= 128) {//compile time
    if (threadIdx. x < 64) { 
      sdata[place] += sdata[place + 64]; } __syncthreads();
       }
  //Instructions are SIMD synchronous within a warp
  //so no need for __syncthreads(), in the last iterations
  if (threadIdx.x < mymin(32u,numThreads/2)) {
    warpReduce<numThreads>(sdata, place);
  }
  // write result for this block to global mem
  if (threadIdx.x == 0){
    g_odata[blockIdx.x] = sdata[0];
  }
}

//after c++14 the template activation will be shorter to write:
//template<typename T, std::enable_if_t<std::is_integral_v<T>, bool> = true>

///finds the nearest upper multiple of the given reference (wit non increments)
template<typename T, 
typename std::enable_if<std::is_integral<T>::value, bool>::type = true>
  inline T nearestUpperMultipleTo(T number, T reference){
    return ((number-1)|(reference-1))+1;
}

///We'll find the ideal number of blocks using the Brent's theorem
size_t getIdealGroups(size_t numberOfElements, size_t runningThreads){
    //nearest upper multiple to the numberof threads
    const size_t nnToGPU=nearestUpperMultipleTo(numberOfElements,runningThreads);
    ///Brent’s theorem says each thread should sum O(log n) elements
    //const size_t elementsPerThread=log(nnToGPU);
    const size_t expectedTotalThreads = ceil(nnToGPU/log(nnToGPU));
    //hence the blocks should have this size:
    const unsigned ngroups = nearestUpperMultipleTo(expectedTotalThreads,runningThreads)/runningThreads;
    return  ngroups;
}


size_t decideThreadsPerBlock(unsigned N, unsigned maxNumThreads=512){
  //this seeks the minimum number of threads to use a sigle block (and end the recursion)
  size_t dim=32;
  for (dim=32;dim<512;dim<<=1){
    if (maxNumThreads < dim) {
      dim >>=1;
      break;
    }
    if( N < dim){
      break;
    }
  }
  return dim;
}

template <typename T>
void callReduction1D (T *g_idata, T *g_odata, const unsigned int len, const unsigned blocks, const unsigned nthreads){
  switch (nthreads) {
  case 512:
    reduction1D<512,T><<<blocks,512,512*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 256:
    reduction1D<256,T><<<blocks,256,256*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 128:
    reduction1D<128,T><<<blocks,128,128*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 64:
    reduction1D<64, T><<<blocks,64,64*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 32:
    reduction1D<32, T><<<blocks,32,32*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  default:
    plumed_merror("Reduction can be called only with 512, 256, 128, 64 or 32 threads.");
  }
}

template <typename T>
void callReductionND (T *g_idata, T *g_odata, const unsigned int len, const dim3 blocks, const unsigned nthreads){
    switch (nthreads) {
  case 512:
    reductionND<512,T><<<blocks,512,512*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 256:
    reductionND<256,T><<<blocks,256,256*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 128:
    reductionND<128,T><<<blocks,128,128*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 64:
    reductionND<64, T><<<blocks,64,64*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  case 32:
    reductionND<32, T><<<blocks,32,32*sizeof(T)>>>(g_idata,g_odata, len);
    break;
  default:
    plumed_merror("Reduction can be called only with 512, 256, 128, 64 or 32 threads.");
  }
}

double reduceScalar(double* cudaScalarAddress, unsigned N, unsigned maxNumThreads){
//we'll proceed to call recursively callreduction1D until N==1:
  double *reduceOut = cudaScalarAddress;
  double *reduceIn;
  while(N>1){
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    reduceIn = reduceOut;
    reduceOut = nullptr;
    auto ngroups=getIdealGroups(N, runningThreads);
    hipFree(reduceOut);
    hipMalloc(&reduceOut,ngroups  * sizeof(double));
    callReduction1D (reduceIn, reduceOut, N, ngroups, runningThreads);
    if (reduceIn != cudaScalarAddress){
      hipFree(reduceIn);
    }
    N=ngroups;
  }
  double toret;
  hipMemcpy(&toret, reduceOut, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(reduceOut);
  return toret;
}

double reduceScalar(memoryHolder<double>& cudaScalarAddress,
 memoryHolder<double>& memoryHelper,
  unsigned N, unsigned maxNumThreads){
//we'll proceed to call recursively callreduction1D until N==1:
  memoryHolder<double>* reduceIn= &memoryHelper;
  memoryHolder<double>* reduceOut =&cudaScalarAddress;
  while(N>1){
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    std::swap(reduceIn,reduceOut);
    auto ngroups=getIdealGroups(N, runningThreads);
    reduceOut->resize(ngroups);
    callReduction1D (reduceIn->getPointer(), reduceOut->getPointer(), N, ngroups, runningThreads);
    N=ngroups;
  }
  double toret;
  reduceOut->copyFromCuda(&toret);
  return toret;
}

std::vector<Vector> reduceNVectors(double* cudaNVectorAddress, unsigned N, unsigned nat, unsigned maxNumThreads){
  double *reduceOut = cudaNVectorAddress;
  double *reduceIn;
  auto dim = nat*3;
  vdbg("InNVectors");
  while(N>1){
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    reduceIn = reduceOut;
    reduceOut = nullptr;
    vdbg(N);
    vdbg(cudaNVectorAddress);
    vdbg(reduceIn);
    vdbg(reduceOut);
    vdbg(runningThreads);
    dim3 ngroups(getIdealGroups(N, runningThreads),dim);
    vdbg(ngroups.x);
    vdbg(ngroups.y);
    hipFree(reduceOut);
    hipMalloc(&reduceOut,ngroups.y* ngroups.x  * sizeof(double));
    vdbg(reduceOut);

    callReductionND (reduceIn, reduceOut, N, ngroups, runningThreads);

    if (reduceIn != cudaNVectorAddress){
      vdbg("Free reduceIn");
      hipFree(reduceIn);
    }
    N=ngroups.x;
  }
  std::vector<Vector> toret(nat);
  hipMemcpy(&toret[0][0], reduceOut, 3*nat*sizeof(double), hipMemcpyDeviceToHost);
  hipFree(reduceOut);
  vdbg(toret[0]);
  return toret;
}

//THIS DOES NOT KEEP THE DATA SAFE
std::vector<Vector> reduceNVectors(memoryHolder<double>& cudaNVectorAddress,
 memoryHolder<double>& memoryHelper, 
unsigned N, unsigned nat, unsigned maxNumThreads){
  memoryHolder<double>* reduceIn= &memoryHelper;
  memoryHolder<double>* reduceOut =&cudaNVectorAddress;
  
  auto dim = nat*3;
  vdbg("InNVectors");
  while(N>1){
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    std::swap(reduceIn,reduceOut);
    dim3 ngroups(getIdealGroups(N, runningThreads),dim);
    reduceOut->resize(ngroups.y* ngroups.x);
    

    callReductionND (reduceIn->getPointer(), reduceOut->getPointer(), N, ngroups, runningThreads);

    N=ngroups.x;
  }
  std::vector<Vector> toret(nat);
  reduceOut->copyFromCuda(&toret[0][0]);
  //hipMemcpy(&toret[0][0], reduceOut, 3*nat*sizeof(double), hipMemcpyDeviceToHost);
  
  vdbg(toret[0]);
  return toret;
}

Vector reduceVector(double* cudaVectorAddress, unsigned N, unsigned maxNumThreads){
///@TODO:This is not tested as now
//we'll proceed to call recursively callreduction1D until N==1:
  double *reduceOut = cudaVectorAddress;
  double *reduceIn;
  while(N>1){
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    reduceIn = reduceOut;
    reduceOut = nullptr;
    dim3 ngroups(getIdealGroups(N, runningThreads),3);
    hipFree(reduceOut);
    hipMalloc(&reduceOut,ngroups.y* ngroups.x  * sizeof(double));
    
    callReductionND (reduceIn, reduceOut, N, ngroups, runningThreads);
        
    if (reduceIn != cudaVectorAddress){
      hipFree(reduceIn);
    }
    N=ngroups.x;
  }
  Vector toret;
  hipMemcpy(&toret[0], reduceOut, 3*sizeof(double), hipMemcpyDeviceToHost);
  hipFree(reduceOut);
  return toret;
}

//#define vdbg(...) std::cerr << std::setw(4) << __LINE__ <<":" << std::setw(20)<< #__VA_ARGS__ << " " << (__VA_ARGS__) <<'\n'
Tensor reduceTensor(memoryHolder<double>&  cudaTensorAddress, 
memoryHolder<double>& memoryHelper, unsigned N, unsigned maxNumThreads){
//we'll proceed to call recursively callreduction1D until N==1:
  memoryHolder<double>* reduceIn= &memoryHelper;
  memoryHolder<double>* reduceOut =&cudaTensorAddress;
  while(N>1){
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    std::swap(reduceIn,reduceOut);
    dim3 ngroups(getIdealGroups(N, runningThreads),9);
    reduceOut->resize(ngroups.y* ngroups.x);
  
    callReductionND (reduceIn->getPointer(), reduceOut->getPointer(),
    N, ngroups, runningThreads);
        
    N=ngroups.x;
  }
  Tensor toret;
  reduceOut->copyFromCuda(&toret[0][0]);
  return toret;
}

DVS::DVS(unsigned nat): deriv(nat){}

template <typename T>
void callReduction1D (T *g_idata, T *g_odata, const unsigned int len,
 const unsigned blocks, const unsigned nthreads,hipStream_t& stream){
  switch (nthreads) {
  case 512:
    reduction1D<512,T><<<blocks,512,512*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 256:
    reduction1D<256,T><<<blocks,256,256*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 128:
    reduction1D<128,T><<<blocks,128,128*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 64:
    reduction1D<64, T><<<blocks,64,64*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 32:
    reduction1D<32, T><<<blocks,32,32*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  default:
    plumed_merror("Reduction can be called only with 512, 256, 128, 64 or 32 threads.");
  }
}

template <typename T>
void callReductionND (T *g_idata, T *g_odata, const unsigned int len,
 const dim3 blocks, const unsigned nthreads,hipStream_t& stream){
    switch (nthreads) {
  case 512:
    reductionND<512,T><<<blocks,512,512*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 256:
    reductionND<256,T><<<blocks,256,256*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 128:
    reductionND<128,T><<<blocks,128,128*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 64:
    reductionND<64, T><<<blocks,64,64*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  case 32:
    reductionND<32, T><<<blocks,32,32*sizeof(T),stream>>>(g_idata,g_odata, len);
    break;
  default:
    plumed_merror("Reduction can be called only with 512, 256, 128, 64 or 32 threads.");
  }
}


DVS reduceDVS(memoryHolder<double>& cudaV,
memoryHolder<double>& cudaT,
memoryHolder<double>& cudaS,
 memoryHolder<double>& memoryHelperV, 
 memoryHolder<double>& memoryHelperT, 
 memoryHolder<double>& memoryHelperS, 
unsigned N, unsigned nat, unsigned maxNumThreads){
  memoryHolder<double>* reduceSIn= &memoryHelperS;
  memoryHolder<double>* reduceSOut =&cudaS;
  memoryHolder<double>* reduceTIn= &memoryHelperT;
  memoryHolder<double>* reduceTOut =&cudaT;
  memoryHolder<double>* reduceVIn= &memoryHelperV;
  memoryHolder<double>* reduceVOut =&cudaV;
  
  auto dim = nat*3;
  vdbg("InNVectors");
  hipStream_t streamV;
  hipStream_t streamT;
  hipStream_t streamS;
  hipStreamCreate(&streamV);
  hipStreamCreate(&streamT);
  hipStreamCreate(&streamS);
  while(N>1){
    vdbg(N);
    size_t runningThreads = decideThreadsPerBlock(N,maxNumThreads);
    std::swap(reduceTIn,reduceTOut);
    std::swap(reduceSIn,reduceSOut);
    std::swap(reduceVIn,reduceVOut);
    unsigned ngroupsS=getIdealGroups(N, runningThreads);
    dim3 ngroupsV(ngroupsS,dim);
    dim3 ngroupsT(ngroupsS,9);
    
    reduceVOut->resize(ngroupsV.y* ngroupsV.x);
    reduceTOut->resize(ngroupsT.y* ngroupsT.x);
    reduceSOut->resize(ngroupsS);
    vdbg("Calls");
    callReductionND (reduceVIn->getPointer(), reduceVOut->getPointer(), N, ngroupsV, runningThreads,streamV);
    vdbg("Calls");
    callReductionND (reduceTIn->getPointer(), reduceTOut->getPointer(), N, ngroupsT, runningThreads,streamT);
    vdbg("Calls");
    callReduction1D (reduceSIn->getPointer(), reduceSOut->getPointer(), N, ngroupsS, runningThreads,streamS);
    
    hipDeviceSynchronize();
    vdbg("End");
    N=ngroupsS;
  }
  DVS toret(nat);
  reduceVOut->copyFromCuda(&toret.deriv[0][0]);
  reduceTOut->copyFromCuda(&toret.virial[0][0]);
  reduceSOut->copyFromCuda(&toret.scalar);
  hipStreamDestroy(streamV);
  hipStreamDestroy(streamT);
  hipStreamDestroy(streamS);
  return toret;
}


} //namespace CUDAHELPERS
} //namespace PLMD

